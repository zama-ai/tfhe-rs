#include "hip/hip_runtime.h"
#include "../polynomial/parameters.cuh"
#include "programmable_bootstrap_cg_multibit.cuh"
#include "programmable_bootstrap_multibit.cuh"
#include "programmable_bootstrap_multibit.h"

bool has_support_to_cuda_programmable_bootstrap_cg_multi_bit(
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t level_count,
    uint32_t num_samples, uint32_t max_shared_memory) {
  return supports_cooperative_groups_on_multibit_programmable_bootstrap<
      uint64_t>(glwe_dimension, polynomial_size, level_count, num_samples,
                max_shared_memory);
}

template <typename Torus>
void cuda_cg_multi_bit_programmable_bootstrap_lwe_ciphertext_vector(
    cuda_stream_t *stream, Torus *lwe_array_out, Torus *lwe_output_indexes,
    Torus *lut_vector, Torus *lut_vector_indexes, Torus *lwe_array_in,
    Torus *lwe_input_indexes, Torus *bootstrapping_key,
    pbs_buffer<Torus, MULTI_BIT> *pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t grouping_factor,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_luts, uint32_t lwe_idx, uint32_t max_shared_memory,
    uint32_t lwe_chunk_size) {

  if (base_log > 64)
    PANIC("Cuda error (multi-bit PBS): base log should be > number of bits in "
          "the ciphertext representation (64)");

  switch (polynomial_size) {
  case 256:
    host_cg_multi_bit_programmable_bootstrap<uint64_t, int64_t,
                                             AmortizedDegree<256>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 512:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<512>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 1024:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<1024>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 2048:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<2048>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 4096:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<4096>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 8192:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<8192>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 16384:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<16384>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

template <typename Torus>
void cuda_multi_bit_programmable_bootstrap_lwe_ciphertext_vector(
    cuda_stream_t *stream, Torus *lwe_array_out, Torus *lwe_output_indexes,
    Torus *lut_vector, Torus *lut_vector_indexes, Torus *lwe_array_in,
    Torus *lwe_input_indexes, Torus *bootstrapping_key,
    pbs_buffer<Torus, MULTI_BIT> *pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t grouping_factor,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_luts, uint32_t lwe_idx, uint32_t max_shared_memory,
    uint32_t lwe_chunk_size) {

  if (base_log > 64)
    PANIC("Cuda error (multi-bit PBS): base log should be > number of bits in "
          "the ciphertext representation (64)");

  switch (polynomial_size) {
  case 256:
    host_multi_bit_programmable_bootstrap<uint64_t, int64_t,
                                          AmortizedDegree<256>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 512:
    host_multi_bit_programmable_bootstrap<Torus, int64_t, AmortizedDegree<512>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 1024:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<1024>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 2048:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<2048>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 4096:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<4096>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 8192:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<8192>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  case 16384:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<16384>>(
        stream, lwe_array_out, lwe_output_indexes, lut_vector,
        lut_vector_indexes, lwe_array_in, lwe_input_indexes, bootstrapping_key,
        pbs_buffer, glwe_dimension, lwe_dimension, polynomial_size,
        grouping_factor, base_log, level_count, num_samples, num_luts, lwe_idx,
        max_shared_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

void cuda_multi_bit_programmable_bootstrap_lwe_ciphertext_vector_64(
    cuda_stream_t *stream, void *lwe_array_out, void *lwe_output_indexes,
    void *lut_vector, void *lut_vector_indexes, void *lwe_array_in,
    void *lwe_input_indexes, void *bootstrapping_key, int8_t *mem_ptr,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t grouping_factor, uint32_t base_log, uint32_t level_count,
    uint32_t num_samples, uint32_t num_luts, uint32_t lwe_idx,
    uint32_t max_shared_memory, uint32_t lwe_chunk_size) {

  pbs_buffer<uint64_t, MULTI_BIT> *buffer =
      (pbs_buffer<uint64_t, MULTI_BIT> *)mem_ptr;

  switch (buffer->pbs_variant) {
  case PBS_VARIANT::CG:
    cuda_cg_multi_bit_programmable_bootstrap_lwe_ciphertext_vector<uint64_t>(
        stream, static_cast<uint64_t *>(lwe_array_out),
        static_cast<uint64_t *>(lwe_output_indexes),
        static_cast<uint64_t *>(lut_vector),
        static_cast<uint64_t *>(lut_vector_indexes),
        static_cast<uint64_t *>(lwe_array_in),
        static_cast<uint64_t *>(lwe_input_indexes),
        static_cast<uint64_t *>(bootstrapping_key), buffer, lwe_dimension,
        glwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case PBS_VARIANT::DEFAULT:
    cuda_multi_bit_programmable_bootstrap_lwe_ciphertext_vector<uint64_t>(
        stream, static_cast<uint64_t *>(lwe_array_out),
        static_cast<uint64_t *>(lwe_output_indexes),
        static_cast<uint64_t *>(lut_vector),
        static_cast<uint64_t *>(lut_vector_indexes),
        static_cast<uint64_t *>(lwe_array_in),
        static_cast<uint64_t *>(lwe_input_indexes),
        static_cast<uint64_t *>(bootstrapping_key), buffer, lwe_dimension,
        glwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported implementation variant.")
  }
}

template <typename Torus, typename STorus>
void scratch_cuda_cg_multi_bit_programmable_bootstrap(
    cuda_stream_t *stream, pbs_buffer<Torus, MULTI_BIT> **buffer,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t level_count, uint32_t grouping_factor,
    uint32_t input_lwe_ciphertext_count, uint32_t max_shared_memory,
    bool allocate_gpu_memory, uint32_t lwe_chunk_size) {

  switch (polynomial_size) {
  case 256:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<256>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 512:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<512>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 1024:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<1024>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 2048:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<2048>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 4096:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<4096>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 8192:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<8192>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 16384:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<16384>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

template <typename Torus, typename STorus>
void scratch_cuda_multi_bit_programmable_bootstrap(
    cuda_stream_t *stream, pbs_buffer<Torus, MULTI_BIT> **buffer,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t level_count, uint32_t grouping_factor,
    uint32_t input_lwe_ciphertext_count, uint32_t max_shared_memory,
    bool allocate_gpu_memory, uint32_t lwe_chunk_size) {

  switch (polynomial_size) {
  case 256:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<256>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 512:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<512>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 1024:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<1024>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 2048:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<2048>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 4096:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<4096>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 8192:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<8192>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 16384:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<16384>>(
        stream, buffer, lwe_dimension, glwe_dimension, polynomial_size,
        level_count, input_lwe_ciphertext_count, grouping_factor,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

void scratch_cuda_multi_bit_programmable_bootstrap_64(
    cuda_stream_t *stream, int8_t **buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t level_count,
    uint32_t grouping_factor, uint32_t input_lwe_ciphertext_count,
    uint32_t max_shared_memory, bool allocate_gpu_memory,
    uint32_t lwe_chunk_size) {

  if (supports_cooperative_groups_on_multibit_programmable_bootstrap<uint64_t>(
          glwe_dimension, polynomial_size, level_count,
          input_lwe_ciphertext_count, max_shared_memory))
    scratch_cuda_cg_multi_bit_programmable_bootstrap<uint64_t, int64_t>(
        stream, (pbs_buffer<uint64_t, MULTI_BIT> **)buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count, grouping_factor,
        input_lwe_ciphertext_count, max_shared_memory, allocate_gpu_memory,
        lwe_chunk_size);
  else
    scratch_cuda_multi_bit_programmable_bootstrap<uint64_t, int64_t>(
        stream, (pbs_buffer<uint64_t, MULTI_BIT> **)buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count, grouping_factor,
        input_lwe_ciphertext_count, max_shared_memory, allocate_gpu_memory,
        lwe_chunk_size);
}

void cleanup_cuda_multi_bit_programmable_bootstrap(cuda_stream_t *stream,
                                                   int8_t **buffer) {
  auto x = (pbs_buffer<uint64_t, MULTI_BIT> *)(*buffer);
  x->release(stream);
}

/**
 * Computes divisors of the product of num_sms (streaming multiprocessors on the
 * GPU) and max_blocks_per_sm (maximum active blocks per SM to launch
 * device_multi_bit_programmable_bootstrap_keybundle) smaller than its square
 * root, based on max_num_pbs. If log2(max_num_pbs) <= 13, selects the first
 * suitable divisor. If greater, calculates an offset as max(1,log2(max_num_pbs)
 * - 13) for additional logic.
 *
 * The value 13 was empirically determined based on memory requirements for
 * benchmarking on an RTX 4090 GPU, balancing performance and resource use.
 */
template <typename Torus, class params>
__host__ uint32_t get_lwe_chunk_size(int gpu_index, uint32_t max_num_pbs,
                                     uint32_t polynomial_size,
                                     uint32_t max_shared_memory) {

  uint64_t full_sm_keybundle =
      get_buffer_size_full_sm_multibit_programmable_bootstrap_keybundle<Torus>(
          polynomial_size);

  int max_blocks_per_sm;
  if (max_shared_memory < full_sm_keybundle)
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_blocks_per_sm,
        device_multi_bit_programmable_bootstrap_keybundle<Torus, params, NOSM>,
        polynomial_size / params::opt, full_sm_keybundle);
  else
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_blocks_per_sm,
        device_multi_bit_programmable_bootstrap_keybundle<Torus, params,
                                                          FULLSM>,
        polynomial_size / params::opt, 0);

  int num_sms = 0;
  check_cuda_error(hipDeviceGetAttribute(
      &num_sms, hipDeviceAttributeMultiprocessorCount, gpu_index));

  int x = num_sms * max_blocks_per_sm;
  int count = 0;

  int divisor = 1;
  int ith_divisor = 0;

#if CUDA_ARCH < 900
  // We pick a smaller divisor on GPUs other than H100, so 256-bit integer
  // multiplication can run
  int log2_max_num_pbs = std::log2(max_num_pbs);
  if (log2_max_num_pbs > 13)
    ith_divisor = log2_max_num_pbs - 11;
#endif

  for (int i = sqrt(x); i >= 1; i--) {
    if (x % i == 0) {
      if (count == ith_divisor) {
        divisor = i;
        break;
      } else {
        count++;
      }
    }
  }

  return divisor;
}

template void scratch_cuda_multi_bit_programmable_bootstrap<uint64_t, int64_t>(
    cuda_stream_t *stream, pbs_buffer<uint64_t, MULTI_BIT> **pbs_buffer,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t level_count, uint32_t grouping_factor,
    uint32_t input_lwe_ciphertext_count, uint32_t max_shared_memory,
    bool allocate_gpu_memory, uint32_t lwe_chunk_size);

template void
cuda_multi_bit_programmable_bootstrap_lwe_ciphertext_vector<uint64_t>(
    cuda_stream_t *stream, uint64_t *lwe_array_out,
    uint64_t *lwe_output_indexes, uint64_t *lut_vector,
    uint64_t *lut_vector_indexes, uint64_t *lwe_array_in,
    uint64_t *lwe_input_indexes, uint64_t *bootstrapping_key,
    pbs_buffer<uint64_t, MULTI_BIT> *pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t grouping_factor,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_luts, uint32_t lwe_idx, uint32_t max_shared_memory,
    uint32_t lwe_chunk_size);

template void
scratch_cuda_cg_multi_bit_programmable_bootstrap<uint64_t, int64_t>(
    cuda_stream_t *stream, pbs_buffer<uint64_t, MULTI_BIT> **pbs_buffer,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t level_count, uint32_t grouping_factor,
    uint32_t input_lwe_ciphertext_count, uint32_t max_shared_memory,
    bool allocate_gpu_memory, uint32_t lwe_chunk_size);

template void
cuda_cg_multi_bit_programmable_bootstrap_lwe_ciphertext_vector<uint64_t>(
    cuda_stream_t *stream, uint64_t *lwe_array_out,
    uint64_t *lwe_output_indexes, uint64_t *lut_vector,
    uint64_t *lut_vector_indexes, uint64_t *lwe_array_in,
    uint64_t *lwe_input_indexes, uint64_t *bootstrapping_key,
    pbs_buffer<uint64_t, MULTI_BIT> *pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t grouping_factor,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_luts, uint32_t lwe_idx, uint32_t max_shared_memory,
    uint32_t lwe_chunk_size);
