#include "hip/hip_runtime.h"
#include "../polynomial/parameters.cuh"
#include "programmable_bootstrap_cg_multibit.cuh"
#include "programmable_bootstrap_multibit.cuh"
#include "programmable_bootstrap_multibit.h"

bool has_support_to_cuda_programmable_bootstrap_cg_multi_bit(
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t level_count,
    uint32_t num_samples, uint32_t max_shared_memory) {
  return supports_cooperative_groups_on_multibit_programmable_bootstrap<
      uint64_t>(glwe_dimension, polynomial_size, level_count, num_samples,
                max_shared_memory);
}

template <typename Torus>
void cuda_cg_multi_bit_programmable_bootstrap_lwe_ciphertext_vector(
    void *stream, uint32_t gpu_index, Torus *lwe_array_out,
    Torus *lwe_output_indexes, Torus *lut_vector, Torus *lut_vector_indexes,
    Torus *lwe_array_in, Torus *lwe_input_indexes, Torus *bootstrapping_key,
    pbs_buffer<Torus, MULTI_BIT> *pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t grouping_factor,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_luts, uint32_t lwe_idx, uint32_t max_shared_memory,
    uint32_t lwe_chunk_size) {

  if (base_log > 64)
    PANIC("Cuda error (multi-bit PBS): base log should be > number of bits in "
          "the ciphertext representation (64)");

  switch (polynomial_size) {
  case 256:
    host_cg_multi_bit_programmable_bootstrap<uint64_t, int64_t,
                                             AmortizedDegree<256>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 512:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<512>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 1024:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<1024>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 2048:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<2048>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 4096:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<4096>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 8192:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<8192>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 16384:
    host_cg_multi_bit_programmable_bootstrap<Torus, int64_t,
                                             AmortizedDegree<16384>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

template <typename Torus>
void cuda_multi_bit_programmable_bootstrap_lwe_ciphertext_vector(
    void *stream, uint32_t gpu_index, Torus *lwe_array_out,
    Torus *lwe_output_indexes, Torus *lut_vector, Torus *lut_vector_indexes,
    Torus *lwe_array_in, Torus *lwe_input_indexes, Torus *bootstrapping_key,
    pbs_buffer<Torus, MULTI_BIT> *pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t grouping_factor,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_luts, uint32_t lwe_idx, uint32_t max_shared_memory,
    uint32_t lwe_chunk_size) {

  if (base_log > 64)
    PANIC("Cuda error (multi-bit PBS): base log should be > number of bits in "
          "the ciphertext representation (64)");

  switch (polynomial_size) {
  case 256:
    host_multi_bit_programmable_bootstrap<uint64_t, int64_t,
                                          AmortizedDegree<256>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 512:
    host_multi_bit_programmable_bootstrap<Torus, int64_t, AmortizedDegree<512>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 1024:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<1024>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 2048:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<2048>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 4096:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<4096>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 8192:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<8192>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  case 16384:
    host_multi_bit_programmable_bootstrap<Torus, int64_t,
                                          AmortizedDegree<16384>>(
        static_cast<hipStream_t>(stream), gpu_index, lwe_array_out,
        lwe_output_indexes, lut_vector, lut_vector_indexes, lwe_array_in,
        lwe_input_indexes, bootstrapping_key, pbs_buffer, glwe_dimension,
        lwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

void cuda_multi_bit_programmable_bootstrap_lwe_ciphertext_vector_64(
    void *stream, uint32_t gpu_index, void *lwe_array_out,
    void *lwe_output_indexes, void *lut_vector, void *lut_vector_indexes,
    void *lwe_array_in, void *lwe_input_indexes, void *bootstrapping_key,
    int8_t *buffer, uint32_t lwe_dimension, uint32_t glwe_dimension,
    uint32_t polynomial_size, uint32_t grouping_factor, uint32_t base_log,
    uint32_t level_count, uint32_t num_samples, uint32_t num_luts,
    uint32_t lwe_idx, uint32_t max_shared_memory, uint32_t lwe_chunk_size) {

  if (supports_cooperative_groups_on_multibit_programmable_bootstrap<uint64_t>(
          glwe_dimension, polynomial_size, level_count, num_samples,
          max_shared_memory))
    cuda_cg_multi_bit_programmable_bootstrap_lwe_ciphertext_vector<uint64_t>(
        stream, gpu_index, static_cast<uint64_t *>(lwe_array_out),
        static_cast<uint64_t *>(lwe_output_indexes),
        static_cast<uint64_t *>(lut_vector),
        static_cast<uint64_t *>(lut_vector_indexes),
        static_cast<uint64_t *>(lwe_array_in),
        static_cast<uint64_t *>(lwe_input_indexes),
        static_cast<uint64_t *>(bootstrapping_key),
        (pbs_buffer<uint64_t, MULTI_BIT> *)buffer, lwe_dimension,
        glwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
  else
    cuda_multi_bit_programmable_bootstrap_lwe_ciphertext_vector<uint64_t>(
        stream, gpu_index, static_cast<uint64_t *>(lwe_array_out),
        static_cast<uint64_t *>(lwe_output_indexes),
        static_cast<uint64_t *>(lut_vector),
        static_cast<uint64_t *>(lut_vector_indexes),
        static_cast<uint64_t *>(lwe_array_in),
        static_cast<uint64_t *>(lwe_input_indexes),
        static_cast<uint64_t *>(bootstrapping_key),
        (pbs_buffer<uint64_t, MULTI_BIT> *)buffer, lwe_dimension,
        glwe_dimension, polynomial_size, grouping_factor, base_log, level_count,
        num_samples, num_luts, lwe_idx, max_shared_memory, lwe_chunk_size);
}

template <typename Torus, typename STorus>
void scratch_cuda_cg_multi_bit_programmable_bootstrap(
    void *stream, uint32_t gpu_index, pbs_buffer<Torus, MULTI_BIT> **buffer,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t level_count,
    uint32_t input_lwe_ciphertext_count, uint32_t max_shared_memory,
    bool allocate_gpu_memory, uint32_t lwe_chunk_size) {

  switch (polynomial_size) {
  case 256:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<256>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, glwe_dimension,
        polynomial_size, level_count, input_lwe_ciphertext_count,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 512:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<512>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, glwe_dimension,
        polynomial_size, level_count, input_lwe_ciphertext_count,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 1024:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<1024>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, glwe_dimension,
        polynomial_size, level_count, input_lwe_ciphertext_count,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 2048:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<2048>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, glwe_dimension,
        polynomial_size, level_count, input_lwe_ciphertext_count,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 4096:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<4096>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, glwe_dimension,
        polynomial_size, level_count, input_lwe_ciphertext_count,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 8192:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<8192>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, glwe_dimension,
        polynomial_size, level_count, input_lwe_ciphertext_count,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  case 16384:
    scratch_cg_multi_bit_programmable_bootstrap<Torus, STorus,
                                                AmortizedDegree<16384>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, glwe_dimension,
        polynomial_size, level_count, input_lwe_ciphertext_count,
        max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

template <typename Torus, typename STorus>
void scratch_cuda_multi_bit_programmable_bootstrap(
    void *stream, uint32_t gpu_index, pbs_buffer<Torus, MULTI_BIT> **buffer,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t level_count, uint32_t grouping_factor,
    uint32_t input_lwe_ciphertext_count, uint32_t max_shared_memory,
    bool allocate_gpu_memory, uint32_t lwe_chunk_size) {

  switch (polynomial_size) {
  case 256:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<256>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count,
        input_lwe_ciphertext_count, grouping_factor, max_shared_memory,
        allocate_gpu_memory, lwe_chunk_size);
    break;
  case 512:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<512>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count,
        input_lwe_ciphertext_count, grouping_factor, max_shared_memory,
        allocate_gpu_memory, lwe_chunk_size);
    break;
  case 1024:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<1024>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count,
        input_lwe_ciphertext_count, grouping_factor, max_shared_memory,
        allocate_gpu_memory, lwe_chunk_size);
    break;
  case 2048:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<2048>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count,
        input_lwe_ciphertext_count, grouping_factor, max_shared_memory,
        allocate_gpu_memory, lwe_chunk_size);
    break;
  case 4096:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<4096>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count,
        input_lwe_ciphertext_count, grouping_factor, max_shared_memory,
        allocate_gpu_memory, lwe_chunk_size);
    break;
  case 8192:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<8192>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count,
        input_lwe_ciphertext_count, grouping_factor, max_shared_memory,
        allocate_gpu_memory, lwe_chunk_size);
    break;
  case 16384:
    scratch_multi_bit_programmable_bootstrap<Torus, STorus,
                                             AmortizedDegree<16384>>(
        static_cast<hipStream_t>(stream), gpu_index, buffer, lwe_dimension,
        glwe_dimension, polynomial_size, level_count,
        input_lwe_ciphertext_count, grouping_factor, max_shared_memory,
        allocate_gpu_memory, lwe_chunk_size);
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

void scratch_cuda_multi_bit_programmable_bootstrap_64(
    void *stream, uint32_t gpu_index, int8_t **buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t level_count,
    uint32_t grouping_factor, uint32_t input_lwe_ciphertext_count,
    uint32_t max_shared_memory, bool allocate_gpu_memory,
    uint32_t lwe_chunk_size) {

  if (supports_cooperative_groups_on_multibit_programmable_bootstrap<uint64_t>(
          glwe_dimension, polynomial_size, level_count,
          input_lwe_ciphertext_count, max_shared_memory))
    scratch_cuda_cg_multi_bit_programmable_bootstrap<uint64_t, int64_t>(
        stream, gpu_index, (pbs_buffer<uint64_t, MULTI_BIT> **)buffer,
        glwe_dimension, polynomial_size, level_count,
        input_lwe_ciphertext_count, max_shared_memory, allocate_gpu_memory,
        lwe_chunk_size);
  else
    scratch_cuda_multi_bit_programmable_bootstrap<uint64_t, int64_t>(
        stream, gpu_index, (pbs_buffer<uint64_t, MULTI_BIT> **)buffer,
        lwe_dimension, glwe_dimension, polynomial_size, level_count,
        grouping_factor, input_lwe_ciphertext_count, max_shared_memory,
        allocate_gpu_memory, lwe_chunk_size);
}

void cleanup_cuda_multi_bit_programmable_bootstrap(void *stream,
                                                   uint32_t gpu_index,
                                                   int8_t **buffer) {
  hipSetDevice(gpu_index);
  auto x = (pbs_buffer<uint64_t, MULTI_BIT> *)(*buffer);
  x->release(static_cast<hipStream_t>(stream), gpu_index);
}

// Returns a chunk size that is not optimal but close to
__host__ uint32_t get_lwe_chunk_size(uint32_t ct_count) {

#if CUDA_ARCH >= 900
  // Tesla H100
  return (ct_count > 10000) ? 30 : 64;
#elif CUDA_ARCH >= 890
  // Tesla RTX4090
  return 8;
#elif CUDA_ARCH >= 800
  // Tesla A100
  return (ct_count > 10000) ? 30 : 45;
#elif CUDA_ARCH >= 700
  // Tesla V100
  return (ct_count > 10000) ? 12 : 18;
#else
  // Generic case
  return (ct_count > 10000) ? 2 : 1;
#endif
}

template void scratch_cuda_multi_bit_programmable_bootstrap<uint64_t, int64_t>(
    void *stream, uint32_t gpu_index,
    pbs_buffer<uint64_t, MULTI_BIT> **pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t level_count,
    uint32_t grouping_factor, uint32_t input_lwe_ciphertext_count,
    uint32_t max_shared_memory, bool allocate_gpu_memory,
    uint32_t lwe_chunk_size);

template void
cuda_multi_bit_programmable_bootstrap_lwe_ciphertext_vector<uint64_t>(
    void *stream, uint32_t gpu_index, uint64_t *lwe_array_out,
    uint64_t *lwe_output_indexes, uint64_t *lut_vector,
    uint64_t *lut_vector_indexes, uint64_t *lwe_array_in,
    uint64_t *lwe_input_indexes, uint64_t *bootstrapping_key,
    pbs_buffer<uint64_t, MULTI_BIT> *pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t grouping_factor,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_luts, uint32_t lwe_idx, uint32_t max_shared_memory,
    uint32_t lwe_chunk_size);

template void
scratch_cuda_cg_multi_bit_programmable_bootstrap<uint64_t, int64_t>(
    void *stream, uint32_t gpu_index,
    pbs_buffer<uint64_t, MULTI_BIT> **pbs_buffer, uint32_t glwe_dimension,
    uint32_t polynomial_size, uint32_t level_count,
    uint32_t input_lwe_ciphertext_count, uint32_t max_shared_memory,
    bool allocate_gpu_memory, uint32_t lwe_chunk_size);

template void
cuda_cg_multi_bit_programmable_bootstrap_lwe_ciphertext_vector<uint64_t>(
    void *stream, uint32_t gpu_index, uint64_t *lwe_array_out,
    uint64_t *lwe_output_indexes, uint64_t *lut_vector,
    uint64_t *lut_vector_indexes, uint64_t *lwe_array_in,
    uint64_t *lwe_input_indexes, uint64_t *bootstrapping_key,
    pbs_buffer<uint64_t, MULTI_BIT> *pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t grouping_factor,
    uint32_t base_log, uint32_t level_count, uint32_t num_samples,
    uint32_t num_luts, uint32_t lwe_idx, uint32_t max_shared_memory,
    uint32_t lwe_chunk_size);
