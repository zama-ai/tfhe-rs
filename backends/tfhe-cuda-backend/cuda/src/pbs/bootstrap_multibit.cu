#include "hip/hip_runtime.h"
#include "../polynomial/parameters.cuh"
#include "bootstrap_fast_multibit.cuh"
#include "bootstrap_multibit.cuh"
#include "bootstrap_multibit.h"

void cuda_multi_bit_pbs_lwe_ciphertext_vector_64(
    cuda_stream_t *stream, void *lwe_array_out, void *lwe_output_indexes,
    void *lut_vector, void *lut_vector_indexes, void *lwe_array_in,
    void *lwe_input_indexes, void *bootstrapping_key, int8_t *pbs_buffer,
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t grouping_factor, uint32_t base_log, uint32_t level_count,
    uint32_t num_samples, uint32_t num_luts, uint32_t lwe_idx,
    uint32_t max_shared_memory, uint32_t lwe_chunk_size) {

  if (base_log > 64)
    PANIC("Cuda error (multi-bit PBS): base log should be > number of bits in "
          "the ciphertext representation (64)");

  switch (polynomial_size) {
  case 256:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<256>>(
            glwe_dimension, level_count, num_samples, max_shared_memory)) {
      host_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<256>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    } else {
      host_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<256>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    }
    break;
  case 512:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<512>>(
            glwe_dimension, level_count, num_samples, max_shared_memory)) {
      host_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<512>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    } else {
      host_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<512>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    }
    break;
  case 1024:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<1024>>(
            glwe_dimension, level_count, num_samples, max_shared_memory)) {
      host_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<1024>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    } else {
      host_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<1024>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    }
    break;
  case 2048:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<2048>>(
            glwe_dimension, level_count, num_samples, max_shared_memory)) {
      host_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<2048>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    } else {
      host_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<2048>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    }
    break;
  case 4096:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<4096>>(
            glwe_dimension, level_count, num_samples, max_shared_memory)) {
      host_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<4096>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    } else {
      host_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<4096>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    }
    break;
  case 8192:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<8192>>(
            glwe_dimension, level_count, num_samples, max_shared_memory)) {
      host_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<8192>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    } else {
      host_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<8192>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    }
    break;
  case 16384:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<16384>>(
            glwe_dimension, level_count, num_samples, max_shared_memory)) {
      host_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<16384>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    } else {
      host_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<16384>>(
          stream, static_cast<uint64_t *>(lwe_array_out),
          static_cast<uint64_t *>(lwe_output_indexes),
          static_cast<uint64_t *>(lut_vector),
          static_cast<uint64_t *>(lut_vector_indexes),
          static_cast<uint64_t *>(lwe_array_in),
          static_cast<uint64_t *>(lwe_input_indexes),
          static_cast<uint64_t *>(bootstrapping_key), pbs_buffer,
          glwe_dimension, lwe_dimension, polynomial_size, grouping_factor,
          base_log, level_count, num_samples, num_luts, lwe_idx,
          max_shared_memory, lwe_chunk_size);
    }
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

void scratch_cuda_multi_bit_pbs_64(
    cuda_stream_t *stream, int8_t **pbs_buffer, uint32_t lwe_dimension,
    uint32_t glwe_dimension, uint32_t polynomial_size, uint32_t level_count,
    uint32_t grouping_factor, uint32_t input_lwe_ciphertext_count,
    uint32_t max_shared_memory, bool allocate_gpu_memory,
    uint32_t lwe_chunk_size) {

  switch (polynomial_size) {
  case 256:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<256>>(
            glwe_dimension, level_count, input_lwe_ciphertext_count,
            max_shared_memory)) {
      scratch_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<256>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    } else {
      scratch_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<256>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    }
    break;
  case 512:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<512>>(
            glwe_dimension, level_count, input_lwe_ciphertext_count,
            max_shared_memory)) {
      scratch_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<512>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    } else {
      scratch_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<512>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    }
    break;
  case 1024:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<1024>>(
            glwe_dimension, level_count, input_lwe_ciphertext_count,
            max_shared_memory)) {
      scratch_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<1024>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    } else {
      scratch_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<1024>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    }
    break;
  case 2048:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<2048>>(
            glwe_dimension, level_count, input_lwe_ciphertext_count,
            max_shared_memory)) {
      scratch_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<2048>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    } else {
      scratch_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<2048>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    }
    break;
  case 4096:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<4096>>(
            glwe_dimension, level_count, input_lwe_ciphertext_count,
            max_shared_memory)) {
      scratch_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<4096>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    } else {
      scratch_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<4096>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    }
    break;
  case 8192:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<8192>>(
            glwe_dimension, level_count, input_lwe_ciphertext_count,
            max_shared_memory)) {
      scratch_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<8192>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    } else {
      scratch_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<8192>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    }
    break;
  case 16384:
    if (verify_cuda_bootstrap_fast_multi_bit_grid_size<uint64_t,
                                                       AmortizedDegree<16384>>(
            glwe_dimension, level_count, input_lwe_ciphertext_count,
            max_shared_memory)) {
      scratch_fast_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<16384>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    } else {
      scratch_multi_bit_pbs<uint64_t, int64_t, AmortizedDegree<16384>>(
          stream, pbs_buffer, lwe_dimension, glwe_dimension, polynomial_size,
          level_count, input_lwe_ciphertext_count, grouping_factor,
          max_shared_memory, allocate_gpu_memory, lwe_chunk_size);
    }
    break;
  default:
    PANIC("Cuda error (multi-bit PBS): unsupported polynomial size. Supported "
          "N's are powers of two"
          " in the interval [256..16384].")
  }
}

void cleanup_cuda_multi_bit_pbs(cuda_stream_t *stream, int8_t **pbs_buffer) {

  // Free memory
  cuda_drop_async(*pbs_buffer, stream);
}

// Pick the best possible chunk size for each GPU
__host__ uint32_t get_lwe_chunk_size(uint32_t lwe_dimension,
                                     uint32_t level_count,
                                     uint32_t glwe_dimension,
                                     uint32_t num_samples) {

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0); // Assuming device 0

  const char *v100Name = "V100"; // Known name of V100 GPU
  const char *a100Name = "A100"; // Known name of A100 GPU
  const char *h100Name = "H100"; // Known name of H100 GPU

  if (std::strstr(deviceProp.name, v100Name) != nullptr) {
    // Tesla V100
    if (num_samples == 1)
      return 60;
    else if (num_samples == 2)
      return 40;
    else if (num_samples <= 4)
      return 20;
    else if (num_samples <= 8)
      return 10;
    else if (num_samples <= 16)
      return 40;
    else if (num_samples <= 32)
      return 27;
    else if (num_samples <= 64)
      return 20;
    else if (num_samples <= 128)
      return 18;
    else if (num_samples <= 256)
      return 16;
    else if (num_samples <= 512)
      return 15;
    else if (num_samples <= 1024)
      return 15;
    else
      return 12;
  } else if (std::strstr(deviceProp.name, a100Name) != nullptr) {
    // Tesla A100
    if (num_samples < 4)
      return 11;
    else if (num_samples < 8)
      return 6;
    else if (num_samples < 16)
      return 13;
    else if (num_samples < 64)
      return 19;
    else if (num_samples < 128)
      return 1;
    else if (num_samples < 512)
      return 19;
    else if (num_samples < 1024)
      return 17;
    else if (num_samples < 8192)
      return 19;
    else if (num_samples < 16384)
      return 12;
    else
      return 9;
  } else if (std::strstr(deviceProp.name, h100Name) != nullptr) {
    // Tesla H100
    return 45;
  }

  // Generic case
  return 1;
}

// Returns a chunk size that is not optimal but close to
__host__ uint32_t get_average_lwe_chunk_size(uint32_t lwe_dimension,
                                             uint32_t level_count,
                                             uint32_t glwe_dimension,
                                             uint32_t ct_count) {

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0); // Assuming device 0

  const char *v100Name = "V100"; // Known name of V100 GPU
  const char *a100Name = "A100"; // Known name of A100 GPU
  const char *h100Name = "H100"; // Known name of H100 GPU

  if (std::strstr(deviceProp.name, v100Name) != nullptr) {
    // Tesla V100
    return (ct_count > 10000) ? 12 : 18;
  } else if (std::strstr(deviceProp.name, a100Name) != nullptr) {
    // Tesla A100
    return (ct_count > 10000) ? 30 : 45;
  } else if (std::strstr(deviceProp.name, h100Name) != nullptr) {
    // Tesla H100
    return (ct_count > 10000) ? 30 : 45;
  }

  // Generic case
  return (ct_count > 10000) ? 2 : 10;
}

// Returns the maximum buffer size required to execute batches up to
// max_input_lwe_ciphertext_count
// todo: Deprecate this function
__host__ uint64_t get_max_buffer_size_multibit_bootstrap(
    uint32_t lwe_dimension, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t level_count, uint32_t max_input_lwe_ciphertext_count) {

  uint64_t max_buffer_size = 0;
  for (uint32_t input_lwe_ciphertext_count = 1;
       input_lwe_ciphertext_count <= max_input_lwe_ciphertext_count;
       input_lwe_ciphertext_count *= 2) {
    max_buffer_size = std::max(
        max_buffer_size,
        get_buffer_size_multibit_bootstrap<uint64_t>(
            glwe_dimension, polynomial_size, level_count,
            input_lwe_ciphertext_count,
            get_lwe_chunk_size(lwe_dimension, level_count, glwe_dimension,
                               input_lwe_ciphertext_count)));
  }

  return max_buffer_size;
}
