#include "expand.cuh"

void cuda_lwe_expand_64(void *const stream, uint32_t gpu_index,
                        void *lwe_array_out, const void *lwe_compact_array_in,
                        uint32_t lwe_dimension, uint32_t num_lwe,
                        const uint32_t *compact_list_length_per_lwe,
                        const void *lwe_compact_input_indexes) {

  switch (lwe_dimension) {
  case 256:
    host_lwe_expand<uint64_t, AmortizedDegree<256>>(
        static_cast<hipStream_t>(stream), gpu_index,
        static_cast<uint64_t *>(lwe_array_out),
        static_cast<const uint64_t *>(lwe_compact_array_in), num_lwe,
        compact_list_length_per_lwe,
        static_cast<const uint32_t *>(lwe_compact_input_indexes));
    break;
  case 512:
    host_lwe_expand<uint64_t, AmortizedDegree<512>>(
        static_cast<hipStream_t>(stream), gpu_index,
        static_cast<uint64_t *>(lwe_array_out),
        static_cast<const uint64_t *>(lwe_compact_array_in), num_lwe,
        compact_list_length_per_lwe,
        static_cast<const uint32_t *>(lwe_compact_input_indexes));
    break;
  case 1024:
    host_lwe_expand<uint64_t, AmortizedDegree<1024>>(
        static_cast<hipStream_t>(stream), gpu_index,
        static_cast<uint64_t *>(lwe_array_out),
        static_cast<const uint64_t *>(lwe_compact_array_in), num_lwe,
        compact_list_length_per_lwe,
        static_cast<const uint32_t *>(lwe_compact_input_indexes));
    break;
  case 2048:
    host_lwe_expand<uint64_t, AmortizedDegree<2048>>(
        static_cast<hipStream_t>(stream), gpu_index,
        static_cast<uint64_t *>(lwe_array_out),
        static_cast<const uint64_t *>(lwe_compact_array_in), num_lwe,
        compact_list_length_per_lwe,
        static_cast<const uint32_t *>(lwe_compact_input_indexes));
    break;
  case 4096:
    host_lwe_expand<uint64_t, AmortizedDegree<4096>>(
        static_cast<hipStream_t>(stream), gpu_index,
        static_cast<uint64_t *>(lwe_array_out),
        static_cast<const uint64_t *>(lwe_compact_array_in), num_lwe,
        compact_list_length_per_lwe,
        static_cast<const uint32_t *>(lwe_compact_input_indexes));
    break;
  case 8192:
    host_lwe_expand<uint64_t, AmortizedDegree<8192>>(
        static_cast<hipStream_t>(stream), gpu_index,
        static_cast<uint64_t *>(lwe_array_out),
        static_cast<const uint64_t *>(lwe_compact_array_in), num_lwe,
        compact_list_length_per_lwe,
        static_cast<const uint32_t *>(lwe_compact_input_indexes));
    break;
  case 16384:
    host_lwe_expand<uint64_t, AmortizedDegree<16384>>(
        static_cast<hipStream_t>(stream), gpu_index,
        static_cast<uint64_t *>(lwe_array_out),
        static_cast<const uint64_t *>(lwe_compact_array_in), num_lwe,
        compact_list_length_per_lwe,
        static_cast<const uint32_t *>(lwe_compact_input_indexes));
    break;
  default:
    PANIC("CUDA error: lwe_dimension not supported."
          "Supported n's are powers of two"
          " in the interval [256..16384].");
    break;
  }
}
