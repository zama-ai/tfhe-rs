#include "integer/comparison.cuh"

void scratch_cuda_integer_radix_comparison_kb_64(
    cuda_stream_t *stream, int8_t **mem_ptr, uint32_t glwe_dimension,
    uint32_t polynomial_size, uint32_t big_lwe_dimension,
    uint32_t small_lwe_dimension, uint32_t ks_level, uint32_t ks_base_log,
    uint32_t pbs_level, uint32_t pbs_base_log, uint32_t grouping_factor,
    uint32_t lwe_ciphertext_count, uint32_t message_modulus,
    uint32_t carry_modulus, PBS_TYPE pbs_type, COMPARISON_TYPE op_type,
    bool allocate_gpu_memory) {

  int_radix_params params(pbs_type, glwe_dimension, polynomial_size,
                          big_lwe_dimension, small_lwe_dimension, ks_level,
                          ks_base_log, pbs_level, pbs_base_log, grouping_factor,
                          message_modulus, carry_modulus);

  switch (op_type) {
  case EQ:
  case NE:
    scratch_cuda_integer_radix_equality_check_kb<uint64_t>(
        stream, (int_comparison_buffer<uint64_t> **)mem_ptr,
        lwe_ciphertext_count, params, op_type, allocate_gpu_memory);
    break;
  case GT:
  case GE:
  case LT:
  case LE:
  case MAX:
  case MIN:
    scratch_cuda_integer_radix_difference_check_kb<uint64_t>(
        stream, (int_comparison_buffer<uint64_t> **)mem_ptr,
        lwe_ciphertext_count, params, op_type, allocate_gpu_memory);
    break;
  }
}

void cuda_comparison_integer_radix_ciphertext_kb_64(
    cuda_stream_t *stream, void *lwe_array_out, void *lwe_array_1,
    void *lwe_array_2, int8_t *mem_ptr, void *bsk, void *ksk,
    uint32_t lwe_ciphertext_count) {

  int_comparison_buffer<uint64_t> *buffer =
      (int_comparison_buffer<uint64_t> *)mem_ptr;
  switch (buffer->op) {
  case EQ:
  case NE:
    host_integer_radix_equality_check_kb<uint64_t>(
        stream, static_cast<uint64_t *>(lwe_array_out),
        static_cast<uint64_t *>(lwe_array_1),
        static_cast<uint64_t *>(lwe_array_2), buffer, bsk,
        static_cast<uint64_t *>(ksk), lwe_ciphertext_count);
    break;
  case GT:
  case GE:
  case LT:
  case LE:
    host_integer_radix_difference_check_kb<uint64_t>(
        stream, static_cast<uint64_t *>(lwe_array_out),
        static_cast<uint64_t *>(lwe_array_1),
        static_cast<uint64_t *>(lwe_array_2), buffer,
        buffer->diff_buffer->operator_f, bsk, static_cast<uint64_t *>(ksk),
        lwe_ciphertext_count);
    break;
  case MAX:
  case MIN:
    host_integer_radix_maxmin_kb<uint64_t>(
        stream, static_cast<uint64_t *>(lwe_array_out),
        static_cast<uint64_t *>(lwe_array_1),
        static_cast<uint64_t *>(lwe_array_2), buffer, bsk,
        static_cast<uint64_t *>(ksk), lwe_ciphertext_count);
    break;
  default:
    PANIC("Cuda error: integer operation not supported")
  }
}

void cleanup_cuda_integer_comparison(cuda_stream_t *stream,
                                     int8_t **mem_ptr_void) {

  int_comparison_buffer<uint64_t> *mem_ptr =
      (int_comparison_buffer<uint64_t> *)(*mem_ptr_void);
  mem_ptr->release(stream);
}
