#include "integer/bitwise_ops.cuh"

void scratch_cuda_integer_radix_bitop_kb_64(
    void *const *streams, uint32_t const *gpu_indexes, uint32_t gpu_count,
    int8_t **mem_ptr, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t big_lwe_dimension, uint32_t small_lwe_dimension, uint32_t ks_level,
    uint32_t ks_base_log, uint32_t pbs_level, uint32_t pbs_base_log,
    uint32_t grouping_factor, uint32_t lwe_ciphertext_count,
    uint32_t message_modulus, uint32_t carry_modulus, PBS_TYPE pbs_type,
    BITOP_TYPE op_type, bool allocate_gpu_memory) {

  int_radix_params params(pbs_type, glwe_dimension, polynomial_size,
                          big_lwe_dimension, small_lwe_dimension, ks_level,
                          ks_base_log, pbs_level, pbs_base_log, grouping_factor,
                          message_modulus, carry_modulus);

  scratch_cuda_integer_radix_bitop_kb<uint64_t>(
      (hipStream_t *)(streams), gpu_indexes, gpu_count,
      (int_bitop_buffer<uint64_t> **)mem_ptr, lwe_ciphertext_count, params,
      op_type, allocate_gpu_memory);
}

void cuda_bitop_integer_radix_ciphertext_kb_64(
    void *const *streams, uint32_t const *gpu_indexes, uint32_t gpu_count,
    void *lwe_array_out, void const *lwe_array_1, void const *lwe_array_2,
    int8_t *mem_ptr, void *const *bsks, void *const *ksks,
    uint32_t lwe_ciphertext_count) {

  host_integer_radix_bitop_kb<uint64_t>(
      (hipStream_t *)(streams), gpu_indexes, gpu_count,
      static_cast<uint64_t *>(lwe_array_out),
      static_cast<const uint64_t *>(lwe_array_1),
      static_cast<const uint64_t *>(lwe_array_2),
      (int_bitop_buffer<uint64_t> *)mem_ptr, bsks, (uint64_t **)(ksks),
      lwe_ciphertext_count);
}

void cleanup_cuda_integer_bitop(void *const *streams,
                                uint32_t const *gpu_indexes, uint32_t gpu_count,
                                int8_t **mem_ptr_void) {

  int_bitop_buffer<uint64_t> *mem_ptr =
      (int_bitop_buffer<uint64_t> *)(*mem_ptr_void);
  mem_ptr->release((hipStream_t *)(streams), gpu_indexes, gpu_count);
}
