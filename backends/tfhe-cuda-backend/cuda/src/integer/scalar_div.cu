#include "scalar_div.cuh"

uint64_t scratch_cuda_integer_unsigned_scalar_div_radix_kb_64(
    void *const *streams, uint32_t const *gpu_indexes, uint32_t gpu_count,
    int8_t **mem_ptr, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t lwe_dimension, uint32_t ks_level, uint32_t ks_base_log,
    uint32_t pbs_level, uint32_t pbs_base_log, uint32_t grouping_factor,
    uint32_t num_blocks, uint32_t message_modulus, uint32_t carry_modulus,
    PBS_TYPE pbs_type, bool allocate_gpu_memory, bool is_divisor_power_of_two,
    bool log2_divisor_exceeds_threshold, bool multiplier_exceeds_threshold,
    uint32_t num_scalar_bits, uint32_t ilog2_divisor, bool allocate_ms_array) {

  int_radix_params params(pbs_type, glwe_dimension, polynomial_size,
                          glwe_dimension * polynomial_size, lwe_dimension,
                          ks_level, ks_base_log, pbs_level, pbs_base_log,
                          grouping_factor, message_modulus, carry_modulus,
                          allocate_ms_array);

  return scratch_integer_unsigned_scalar_div_radix<uint64_t>(
      (hipStream_t *)(streams), gpu_indexes, gpu_count, params,
      (int_unsigned_scalar_div_mem<uint64_t> **)mem_ptr, num_blocks,
      allocate_gpu_memory, is_divisor_power_of_two,
      log2_divisor_exceeds_threshold, multiplier_exceeds_threshold,
      num_scalar_bits, ilog2_divisor);
}

void cuda_integer_unsigned_scalar_div_radix_kb_64(
    void *const *streams, uint32_t const *gpu_indexes, uint32_t gpu_count,
    CudaRadixCiphertextFFI *numerator_ct, int8_t *mem_ptr, void *const *ksks,
    uint64_t const *decomposed_scalar, uint64_t const *has_at_least_one_set,
    const CudaModulusSwitchNoiseReductionKeyFFI *ms_noise_reduction_key,
    void *const *bsks, uint32_t num_scalars, bool multiplier_exceeds_threshold,
    bool is_divisor_power_of_two, bool log2_divisor_exceeds_threshold,
    uint32_t ilog2_divisor, uint64_t shift_pre, uint32_t shift_post,
    uint64_t rhs) {

  host_integer_unsigned_scalar_div_radix<uint64_t>(
      (hipStream_t *)streams, gpu_indexes, gpu_count, numerator_ct,
      (int_unsigned_scalar_div_mem<uint64_t> *)mem_ptr, (uint64_t **)ksks,
      decomposed_scalar, has_at_least_one_set, ms_noise_reduction_key, bsks,
      num_scalars, multiplier_exceeds_threshold, is_divisor_power_of_two,
      log2_divisor_exceeds_threshold, ilog2_divisor, shift_pre, shift_post,
      rhs);
}

void cleanup_cuda_integer_unsigned_scalar_div_radix_kb_64(
    void *const *streams, uint32_t const *gpu_indexes, uint32_t gpu_count,
    int8_t **mem_ptr_void) {

  int_unsigned_scalar_div_mem<uint64_t> *mem_ptr =
      (int_unsigned_scalar_div_mem<uint64_t> *)(*mem_ptr_void);

  mem_ptr->release((hipStream_t *)streams, gpu_indexes, gpu_count);
}
