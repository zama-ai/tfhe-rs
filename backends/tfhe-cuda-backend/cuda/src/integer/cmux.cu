#include "integer/cmux.cuh"

void scratch_cuda_integer_radix_cmux_kb_64(
    void *const *streams, uint32_t const *gpu_indexes, uint32_t gpu_count,
    int8_t **mem_ptr, uint32_t glwe_dimension, uint32_t polynomial_size,
    uint32_t big_lwe_dimension, uint32_t small_lwe_dimension, uint32_t ks_level,
    uint32_t ks_base_log, uint32_t pbs_level, uint32_t pbs_base_log,
    uint32_t grouping_factor, uint32_t lwe_ciphertext_count,
    uint32_t message_modulus, uint32_t carry_modulus, PBS_TYPE pbs_type,
    bool allocate_gpu_memory) {

  int_radix_params params(pbs_type, glwe_dimension, polynomial_size,
                          big_lwe_dimension, small_lwe_dimension, ks_level,
                          ks_base_log, pbs_level, pbs_base_log, grouping_factor,
                          message_modulus, carry_modulus);

  std::function<uint64_t(uint64_t)> predicate_lut_f =
      [](uint64_t x) -> uint64_t { return x == 1; };

  scratch_cuda_integer_radix_cmux_kb<uint64_t>(
      (hipStream_t *)(streams), gpu_indexes, gpu_count,
      (int_cmux_buffer<uint64_t> **)mem_ptr, predicate_lut_f,
      lwe_ciphertext_count, params, allocate_gpu_memory);
}

void cuda_cmux_integer_radix_ciphertext_kb_64(
    void *const *streams, uint32_t const *gpu_indexes, uint32_t gpu_count,
    void *lwe_array_out, void const *lwe_condition, void const *lwe_array_true,
    void const *lwe_array_false, int8_t *mem_ptr, void *const *bsks,
    void *const *ksks, uint32_t lwe_ciphertext_count) {

  host_integer_radix_cmux_kb<uint64_t>(
      (hipStream_t *)(streams), gpu_indexes, gpu_count,
      static_cast<uint64_t *>(lwe_array_out),
      static_cast<const uint64_t *>(lwe_condition),
      static_cast<const uint64_t *>(lwe_array_true),
      static_cast<const uint64_t *>(lwe_array_false),
      (int_cmux_buffer<uint64_t> *)mem_ptr, bsks, (uint64_t **)(ksks),

      lwe_ciphertext_count);
}

void cleanup_cuda_integer_radix_cmux(void *const *streams,
                                     uint32_t const *gpu_indexes,
                                     uint32_t gpu_count,
                                     int8_t **mem_ptr_void) {

  int_cmux_buffer<uint64_t> *mem_ptr =
      (int_cmux_buffer<uint64_t> *)(*mem_ptr_void);
  mem_ptr->release((hipStream_t *)(streams), gpu_indexes, gpu_count);
}
