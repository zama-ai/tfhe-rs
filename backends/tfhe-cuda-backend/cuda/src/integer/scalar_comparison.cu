#include "hip/hip_runtime.h"
#include "integer/scalar_comparison.cuh"

#include <iostream>
#include <utility> // for std::pair

std::pair<bool, bool> get_invert_flags(COMPARISON_TYPE compare) {
  bool invert_operands;
  bool invert_subtraction_result;

  switch (compare) {
  case COMPARISON_TYPE::LT:
    invert_operands = false;
    invert_subtraction_result = false;
    break;
  case COMPARISON_TYPE::LE:
    invert_operands = true;
    invert_subtraction_result = true;
    break;
  case COMPARISON_TYPE::GT:
    invert_operands = true;
    invert_subtraction_result = false;
    break;
  case COMPARISON_TYPE::GE:
    invert_operands = false;
    invert_subtraction_result = true;
    break;
  default:
    PANIC("Cuda error: invalid comparison type")
  }

  return {invert_operands, invert_subtraction_result};
}

void cuda_scalar_comparison_ciphertext_64(
    CudaStreamsFFI streams, CudaRadixCiphertextFFI *lwe_array_out,
    CudaRadixCiphertextFFI const *lwe_array_in, void const *scalar_blocks,
    void const *h_scalar_blocks, int8_t *mem_ptr, void *const *bsks,
    void *const *ksks, uint32_t num_scalar_blocks) {

  // The output ciphertext might be a boolean block or a radix ciphertext
  // depending on the case (eq/gt vs max/min) so the amount of blocks to
  // consider for calculation is the one of the input
  auto num_radix_blocks = lwe_array_in->num_radix_blocks;
  int_comparison_buffer<uint64_t> *buffer =
      (int_comparison_buffer<uint64_t> *)mem_ptr;
  switch (buffer->op) {
  case EQ:
  case NE:
    host_scalar_equality_check<uint64_t>(
        CudaStreams(streams), lwe_array_out, lwe_array_in,
        static_cast<const uint64_t *>(scalar_blocks), buffer, bsks,
        (uint64_t **)(ksks), num_radix_blocks, num_scalar_blocks);
    break;
  case GT:
  case GE:
  case LT:
  case LE:
    if (num_radix_blocks % 2 != 0 && num_radix_blocks != 1)
      PANIC("Cuda error (scalar comparisons): the number of radix blocks has "
            "to be even or equal to 1.")
    host_scalar_difference_check<uint64_t>(
        CudaStreams(streams), lwe_array_out, lwe_array_in,
        static_cast<const uint64_t *>(scalar_blocks),
        static_cast<const uint64_t *>(h_scalar_blocks), buffer,
        buffer->diff_buffer->operator_f, bsks, (uint64_t **)(ksks),
        num_radix_blocks, num_scalar_blocks);
    break;
  case MAX:
  case MIN:
    if (lwe_array_in->num_radix_blocks % 2 != 0)
      PANIC("Cuda error (scalar max/min): the number of radix blocks has to be "
            "even.")
    host_scalar_maxmin<uint64_t>(
        CudaStreams(streams), lwe_array_out, lwe_array_in,
        static_cast<const uint64_t *>(scalar_blocks),
        static_cast<const uint64_t *>(h_scalar_blocks), buffer, bsks,
        (uint64_t **)(ksks), num_radix_blocks, num_scalar_blocks);
    break;
  default:
    PANIC("Cuda error: integer operation not supported")
  }
}
