#include "device.h"
#include "helper.h"
#include <mutex>

int cuda_setup_multi_gpu() {
  int num_gpus = cuda_get_number_of_gpus();
  if (num_gpus == 0)
    PANIC("GPU error: the number of GPUs should be > 0.")
  if (num_gpus > 1) {
    int can_access_peer_ij;
    int can_access_peer_ji;
    for (int i = 1; i < num_gpus; i++) {
      check_cuda_error(hipDeviceCanAccessPeer(&can_access_peer_ij, i, 0));
      check_cuda_error(hipDeviceCanAccessPeer(&can_access_peer_ji, 0, i));
      if (can_access_peer_ij && can_access_peer_ji) {
        hipMemPool_t mempool;
        hipMemAccessDesc desc = {};

        // Enable P2P Access and mempool access
        check_cuda_error(hipSetDevice(i));
        check_cuda_error(hipDeviceGetDefaultMemPool(&mempool, i));
        desc.location.type = hipMemLocationTypeDevice;
        desc.location.id = 0;
        desc.flags = hipMemAccessFlagsProtReadWrite;
        check_cuda_error(
            hipMemPoolSetAccess(mempool, &desc, 1 /* numDescs */));
        check_cuda_error_ignore_specific(hipDeviceEnablePeerAccess(0, 0),
                                         hipErrorPeerAccessAlreadyEnabled);

        check_cuda_error(hipSetDevice(0));
        check_cuda_error(hipDeviceGetDefaultMemPool(&mempool, 0));
        desc.location.type = hipMemLocationTypeDevice;
        desc.location.id = i;
        desc.flags = hipMemAccessFlagsProtReadWrite;
        check_cuda_error(
            hipMemPoolSetAccess(mempool, &desc, 1 /* numDescs */));
        check_cuda_error_ignore_specific(hipDeviceEnablePeerAccess(i, 0),
                                         hipErrorPeerAccessAlreadyEnabled);

      } else {
        PANIC("Multi GPU error: all GPUs should have peer access to GPU 0")
      }
    }
  }
  return num_gpus;
}

void cuda_cleanup_multi_gpu() {

  int num_gpus = cuda_get_number_of_gpus();
  if (num_gpus == 0)
    PANIC("GPU error: the number of GPUs should be > 0.")
  if (num_gpus > 1) {
    int can_access_peer_ij;
    int can_access_peer_ji;
    for (int i = 1; i < num_gpus; i++) {
      check_cuda_error(hipDeviceCanAccessPeer(&can_access_peer_ij, i, 0));
      check_cuda_error(hipDeviceCanAccessPeer(&can_access_peer_ji, 0, i));
      if (can_access_peer_ij && can_access_peer_ji) {
        //// Disable access to memory pool
        hipMemPool_t mempool;
        hipDeviceGetDefaultMemPool(&mempool, i);
        hipMemAccessDesc desc = {};
        desc.location.type = hipMemLocationTypeDevice;
        desc.location.id = 0;
        desc.flags = hipMemAccessFlagsProtNone;
        hipMemPoolSetAccess(mempool, &desc, 1 /* numDescs */);

        hipDeviceGetDefaultMemPool(&mempool, 0);
        desc.location.type = hipMemLocationTypeDevice;
        desc.location.id = i;
        desc.flags = hipMemAccessFlagsProtNone;
        hipMemPoolSetAccess(mempool, &desc, 1 /* numDescs */);
        //  Disable P2P Access
        hipSetDevice(i);
        hipDeviceDisablePeerAccess(0);
        hipSetDevice(0);
        hipDeviceDisablePeerAccess(i);
      } else {
        PANIC("Multi GPU error: all GPUs should have peer access to GPU 0")
      }
    }
  }
}

int get_num_inputs_on_gpu(int total_num_inputs, int gpu_index, int gpu_count) {

  int num_inputs = 0;
  // If there are fewer inputs than GPUs, not all GPUs are active and each
  // active GPU handles 1 input
  if (gpu_count > total_num_inputs) {
    if (gpu_index <= total_num_inputs - 1)
      num_inputs = 1;
  } else {
    // If there are more inputs than GPUs, all GPUs are active and compute over
    // a chunk of the total inputs. The chunk size is smaller on the last GPU.
    num_inputs =
        total_num_inputs / gpu_count + (total_num_inputs % gpu_count != 0);
    if (gpu_index == gpu_count - 1)
      num_inputs = total_num_inputs - (gpu_count - 1) * num_inputs;
  }
  return num_inputs;
}
